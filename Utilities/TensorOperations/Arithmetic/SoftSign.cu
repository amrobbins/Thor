#include "hip/hip_runtime.h"
#include "SoftSign.h"

/**
 * x / (abs(x) + 1)
 */
__global__ void softSign(half *featureOut, half *featureIn, int numElements) {
    int element = blockIdx.x * 1024 + (4 * threadIdx.x);

    if (element >= numElements)
        return;

    double *featureIn_half_4 = (double *)featureIn;
    double finBuffer_half_4[1];
    finBuffer_half_4[0] = featureIn_half_4[element / 4];
    half *finBuffer = (half *)finBuffer_half_4;
    half foutBuffer[4];

    const half one[2] = {(half)1.0f, (half)1.0f};
    ((half2 *)foutBuffer)[0] = __h2div(((half2 *)finBuffer)[0], __hadd2(__habs2(((half2 *)finBuffer)[0]), one);
    ((half2 *)foutBuffer)[1] = __h2div(((half2 *)finBuffer)[0], __hadd2(__habs2(((half2 *)finBuffer)[0]), one);

    double *fout_half_4 = (double *)foutBuffer;
    double *featureOut_half_4 = (double *)featureOut;
    featureOut_half_4[element / 4] = fout_half_4[0];
}

/**
 * d/dx(x/abs(x + 1)) = (abs(x + 1) - x sgn(x + 1))/(x + 1)^2
 * Since this function is discontinuous, the derivative as assigned to the values below near the discontinuity:
 * (-1.45, -1.0]: -5.0
 * (-1.0, -0.55): 5.0
 */
__global__ void softSignBackward(half *errorOut, half *featureIn, half *errorIn, int numElements) {
    const half[2] one = {half(1.0f), half(1.0f)};

    int element = blockIdx.x * 1024 + (4 * threadIdx.x);

    if (element >= numElements)
        return;

    double *featureIn_half_4 = (double *)featureIn;
    double featureInBuffer_half_4[1];
    featureInBuffer_half_4[0] = featureIn_half_4[element / 4];
    half *featureInBuffer = (half *)featureInBuffer_half_4;

    double *errorIn_half_4 = (double *)errorIn;
    double errorInBuffer_half_4[1];
    errorInBuffer_half_4[0] = errorIn_half_4[element / 4];
    half *errorInBuffer = (half *)errorInBuffer_half_4;
    half errorOutBuffer[4];

    half foutBuffer[4];
    half xPlusOne[4];
    half absXPlusOne[4];
    half signXPlusOne[4];
    half xPlusOneSquared[4];
    half xSignXPlusOne[4];
    half absXplusOneSquaredMinusXSign[4];
    half fullDerivative[4];

    ((half2 *)xPlusOne)[0] = __hadd2(((half2 *)finBuffer)[0], ((half2 *)one)[0]);
    ((half2 *)absXPlusOne)[0] = __habs2(((half2 *)xPlusOne)[0]);
    ((half2 *)signXPlusOne)[0] = __hdiv2(((half2 *)xPlusOne)[0], ((half2 *)(absXPlusOne)[0]);
    ((half2 *)xPlusOneSquared)[0] = __hmul2(((half2 *)xPlusOne)[0], ((half2 *)xPlusOne)[0]);
    ((half2 *)xSignXPlusOne)[0] = __hmul2(((half2 *)finBuffer)[0], ((half2 *)signXPlusOne)[0]);
    ((half2 *)absXplusOneSquaredMinusXSign)[0] = __hsub2(((half2 *)absXPlusOne)[0], ((half2 *)xSignXPlusOne)[0]);
    ((half2 *)fullDerivative)[0] = __hdiv2(((half2 *)absXplusOneSquaredMinusXSign)[0], ((half2 *)xPlusOneSquared)[0]);
    ((half2 *)foutBuffer)[0] = __hmul2(((half2 *)errorInBuffer)[0], ((half2 *)fullDerivative)[0]);

    ((half2 *)xPlusOne)[1] = __hadd2(((half2 *)finBuffer)[1], ((half2 *)one)[1]);
    ((half2 *)absXPlusOne)[1] = __habs2(((half2 *)xPlusOne)[1]);
    ((half2 *)signXPlusOne)[1] = __hdiv2(((half2 *)xPlusOne)[1], ((half2 *)(absXPlusOne)[1]);
    ((half2 *)xPlusOneSquared)[1] = __hmul2(((half2 *)xPlusOne)[1], ((half2 *)xPlusOne)[1]);
    ((half2 *)xSignXPlusOne)[1] = __hmul2(((half2 *)finBuffer)[1], ((half2 *)signXPlusOne)[1]);
    ((half2 *)absXplusOneSquaredMinusXSign)[1] = __hsub2(((half2 *)absXPlusOne)[1], ((half2 *)xSignXPlusOne)[1]);
    ((half2 *)fullDerivative)[1] = __hdiv2(((half2 *)absXplusOneSquaredMinusXSign)[1], ((half2 *)xPlusOneSquared)[1]);
    ((half2 *)foutBuffer)[1] = __hmul2(((half2 *)errorInBuffer)[1], ((half2 *)fullDerivative)[1]);

    // Check each for nearness to the discontinuity
    half negativeFive = (half)-5.0f;
    half five = (half)5.0f;
    half negativeOnePointFourFive = (half)-1.45f;
    half negativeOne = (half)-1.0f;
    half negativePointFiveFive = (half)-0.55f;

    if(finBuffer[0] > negativeOnePointFourFive && finBuffer[0] <= negativeOne)
        foutBuffer[0] = __hmul(errorInBuffer[0], negativeFive);
    else if(finBuffer[0] > negativeOne && finBuffer[0] < negativePointFiveFive)
        foutBuffer[0] = __hmul(errorInBuffer[0], five);

    if(finBuffer[1] > negativeOnePointFourFive && finBuffer[1] <= negativeOne)
        foutBuffer[1] = __hmul(errorInBuffer[1], negativeFive);
    else if(finBuffer[1] > negativeOne && finBuffer[1] < negativePointFiveFive)
        foutBuffer[1] = __hmul(errorInBuffer[1], five);

    if(finBuffer[2] > negativeOnePointFourFive && finBuffer[2] <= negativeOne)
        foutBuffer[2] = __hmul(errorInBuffer[2], negativeFive);
    else if(finBuffer[2] > negativeOne && finBuffer[2] < negativePointFiveFive)
        foutBuffer[2] = __hmul(errorInBuffer[2], five);

    if(finBuffer[3] > negativeOnePointFourFive && finBuffer[3] <= negativeOne)
        foutBuffer[3] = __hmul(errorInBuffer[3], negativeFive);
    else if(finBuffer[3] > negativeOne && finBuffer[3] < negativePointFiveFive)
        foutBuffer[3] = __hmul(errorInBuffer[3], five);

    double *errorOutBuffer_half_4 = (double *)errorOutBuffer;
    double *errorOut_half_4 = (double *)errorOut;
    errorOut_half_4[element / 4] = errorOutBuffer_half_4[0];
}

void launchSoftSign(half *featureOut_d, half *featureIn_d, int numElements, Stream stream) {
    dim3 blockSize(256);
    dim3 gridSize((numElements + 1023) / 1024);
    ScopedGpu scopedGpu(stream.getGpuNum());
    softSign<<<gridSize, blockSize, 0, stream>>>(featureOut_d, featureIn_d, numElements);
}

void launchSoftSignBackward(half *errorOut_d, half *featureIn_d, half *errorIn_d, int numElements, Stream stream) {
    dim3 blockSize(256);
    dim3 gridSize((numElements + 1023) / 1024);
    ScopedGpu scopedGpu(stream.getGpuNum());
    softSignBackward<<<gridSize, blockSize, 0, stream>>>(errorOut_d, featureIn_d, errorIn_d, numElements);
}
