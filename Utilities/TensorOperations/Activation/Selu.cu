#include "hip/hip_runtime.h"
#include "Selu.h"

constexpr float SCALE_ALPHA = 1.758099326f;
constexpr float SCALE = 1.05070098f;

/**
 * scale * x when x >= 0
 * scale * alpha * (exp(x) - 1) when x < 0
 * where scale = 1.05070098 and alpha = 1.67326324 are pre-set values
 */
__global__ void selu(half *featureOut, half *featureIn, int numElements) {
    int element = blockIdx.x * 1024 + (4 * threadIdx.x);

    if (element >= numElements)
        return;

    double *featureIn_half_4 = (double *)featureIn;
    double finBuffer_half_4[1];
    finBuffer_half_4[0] = featureIn_half_4[element / 4];
    half *finBuffer = (half *)finBuffer_half_4;
    half foutBuffer[4];

    float fin;
    float fout;

    fin = (float)finBuffer[0];
    if (fin >= 0.0f)
        fout = SCALE * fin;
    else
        fout = SCALE_ALPHA * (expf(fin) - 1.0f);
    foutBuffer[0] = (half)fout;

    element += 1;
    if (element < numElements) {
        fin = finBuffer[1];
        if (fin >= 0.0f)
            fout = SCALE * fin;
        else
            fout = SCALE_ALPHA * (expf(fin) - 1.0f);
        foutBuffer[1] = (half)fout;
    }

    element += 1;
    if (element < numElements) {
        fin = finBuffer[2];
        if (fin >= 0.0f)
            fout = SCALE * fin;
        else
            fout = SCALE_ALPHA * (expf(fin) - 1.0f);
        foutBuffer[2] = (half)fout;
    }

    element += 1;
    if (element < numElements) {
        fin = finBuffer[3];
        if (fin >= 0.0f)
            fout = SCALE * fin;
        else
            fout = SCALE_ALPHA * (expf(fin) - 1.0f);
        foutBuffer[3] = (half)fout;
    }

    double *fout_half_4 = (double *)foutBuffer;
    double *featureOut_half_4 = (double *)featureOut;
    featureOut_half_4[element / 4] = fout_half_4[0];
}

/**
 * d/dx(x) = scale when x >= 0
 * d/dx(alpha * (exp(x) - 1)) = scale * alpha * exp(x) when x < 0
 * where scale = 1.05070098 and alpha = 1.67326324 are pre-set values
 */
__global__ void seluBackward(half *errorOut, half *featureIn, half *errorIn, int numElements) {
    int element = blockIdx.x * 1024 + (4 * threadIdx.x);

    if (element >= numElements)
        return;

    const half zero = half(0.0f);
    float fin;
    float ein;
    float eout;

    double *featureIn_half_4 = (double *)featureIn;
    double featureInBuffer_half_4[1];
    featureInBuffer_half_4[0] = featureIn_half_4[element / 4];
    half *featureInBuffer = (half *)featureInBuffer_half_4;

    double *errorIn_half_4 = (double *)errorIn;
    double errorInBuffer_half_4[1];
    errorInBuffer_half_4[0] = errorIn_half_4[element / 4];
    half *errorInBuffer = (half *)errorInBuffer_half_4;
    half errorOutBuffer[4];

    fin = featureInBuffer[0];
    ein = errorInBuffer[0];
    if (fin >= 0.0f)
        eout = SCALE * ein;
    else
        eout = SCALE_ALPHA * ein * expf(fin);
    errorOutBuffer[0] = (half)eout;

    element += 1;
    if (element < numElements) {
        fin = featureInBuffer[1];
        ein = errorInBuffer[1];
        if (fin >= 0.0f)
            eout = SCALE * ein;
        else
            eout = SCALE_ALPHA * ein * expf(fin);
        errorOutBuffer[1] = (half)eout;
    }

    element += 1;
    if (element < numElements) {
        fin = featureInBuffer[2];
        ein = errorInBuffer[2];
        if (fin >= 0.0f)
            eout = SCALE * ein;
        else
            eout = SCALE_ALPHA * ein * expf(fin);
        errorOutBuffer[2] = (half)eout;
    }

    element += 1;
    if (element < numElements) {
        fin = featureInBuffer[3];
        ein = errorInBuffer[3];
        if (fin >= 0.0f)
            eout = SCALE * ein;
        else
            eout = SCALE_ALPHA * ein * expf(fin);
        errorOutBuffer[3] = (half)eout;
    }

    double *errorOutBuffer_half_4 = (double *)errorOutBuffer;
    double *errorOut_half_4 = (double *)errorOut;
    errorOut_half_4[element / 4] = errorOutBuffer_half_4[0];
}

void launchSelu(half *featureOut_d, half *featureIn_d, int numElements, Stream stream) {
    dim3 blockSize(min(256, numElements));
    dim3 gridSize((numElements + 1023) / 1024);
    ScopedGpu scopedGpu(stream.getGpuNum());
    selu<<<gridSize, blockSize, 0, stream>>>(featureOut_d, featureIn_d, numElements);
}

void launchSeluBackward(half *errorOut_d, half *featureIn_d, half *errorIn_d, int numElements, Stream stream) {
    dim3 blockSize(min(256, numElements));
    dim3 gridSize((numElements + 1023) / 1024);
    ScopedGpu scopedGpu(stream.getGpuNum());
    seluBackward<<<gridSize, blockSize, 0, stream>>>(errorOut_d, featureIn_d, errorIn_d, numElements);
}
