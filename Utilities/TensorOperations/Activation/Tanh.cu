#include "hip/hip_runtime.h"
#include "Tanh.h"

__global__ void tanh(half *dest, half *source, int numElements) {
    int element = blockIdx.x * 1024 + (4 * threadIdx.x);

    if (element >= numElements)
        return;

    double *source_half_4 = (double *)source;
    double finBuffer_half_4[1];
    finBuffer_half_4[0] = source_half_4[element / 4];
    half *finBuffer = (half *)finBuffer_half_4;
    half foutBuffer[4];

    foutBuffer[0] = dest[element] = (half)(tanhf(finBuffer[0]));

    element += 1;
    if (element < numElements)
        foutBuffer[1] = dest[element] = (half)(tanhf(finBuffer[1]));

    element += 1;
    if (element < numElements)
        foutBuffer[2] = dest[element] = (half)(tanhf(finBuffer[2]));

    element += 1;
    if (element < numElements)
        foutBuffer[3] = dest[element] = (half)(tanhf(finBuffer[3]));

    double *fout_half_4 = (double *)foutBuffer;
    double *dest_half_4 = (double *)dest;
    dest_half_4[element / 4] = fout_half_4[0];
}

__global__ void tanhBackward(half *errorOut, half *featureIn, half *errorIn, int numElements) {
    int element = blockIdx.x * 1024 + (4 * threadIdx.x);

    if (element >= numElements)
        return;

    float coshx;
    float fin;
    float ein;
    float eout;

    double *featureIn_half_4 = (double *)featureIn;
    double featureInBuffer_half_4[1];
    featureInBuffer_half_4[0] = featureIn_half_4[element / 4];
    half *featureInBuffer = (half *)featureInBuffer_half_4;

    double *errorIn_half_4 = (double *)errorIn;
    double errorInBuffer_half_4[1];
    errorInBuffer_half_4[0] = errorIn_half_4[element / 4];
    half *errorInBuffer = (half *)errorInBuffer_half_4;

    half errorOutBuffer[4];

    fin = (float)featureInBuffer[0];
    ein = (float)errorInBuffer[0];
    coshx = coshf(fin);
    eout = ein / (coshx * coshx);
    errorOutBuffer[0] = (half)eout;

    element += 1;
    if (element < numElements) {
        fin = (float)featureInBuffer[1];
        ein = (float)errorInBuffer[1];
        coshx = coshf(fin);
        eout = ein / (coshx * coshx);
        errorOutBuffer[1] = (half)eout;
    }

    element += 1;
    if (element < numElements) {
        fin = (float)featureInBuffer[2];
        ein = (float)errorInBuffer[2];
        coshx = coshf(fin);
        eout = ein / (coshx * coshx);
        errorOutBuffer[2] = (half)eout;
    }

    element += 1;
    if (element < numElements) {
        fin = (float)featureInBuffer[3];
        ein = (float)errorInBuffer[3];
        coshx = coshf(fin);
        eout = ein / (coshx * coshx);
        errorOutBuffer[3] = (half)eout;
    }

    double *errorOutBuffer_half_4 = (double *)errorOutBuffer;
    double *errorOut_half_4 = (double *)errorOut;
    errorOut_half_4[element / 4] = errorOutBuffer_half_4[0];
}

void launchTanh(half *dest_d, half *source_d, int numElements, Stream stream) {
    dim3 blockSize(256);
    dim3 gridSize((numElements + 1023) / 1024);
    ScopedGpu scopedGpu(stream.getGpuNum());
    tanh<<<gridSize, blockSize, 0, stream>>>(dest_d, source_d, numElements);
}

void launchTanhBackward(half *errorOut_d, half *featureIn_d, half *errorIn_d, int numElements, Stream stream) {
    dim3 blockSize(256);
    dim3 gridSize((numElements + 1023) / 1024);
    ScopedGpu scopedGpu(stream.getGpuNum());
    tanhBackward<<<gridSize, blockSize, 0, stream>>>(errorOut_d, featureIn_d, errorIn_d, numElements);
}
