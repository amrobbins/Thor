#include "hip/hip_runtime.h"
#include "MeanSquaredError.h"

using namespace std;

/**
 * MSE(batch_of_predictions, batch_of_labels) = (1/batchSize) * (batch_of_predictions - batch_of_labels)^2
 *
 * Where the subtraction and squaring are performed element-wise.
 *
 * When there are multiple predictions, there must be the corresponding number of labels.
 * This is enforced via assertion, the loss layer will not run if the size is not correct.
 * In that case the computation goes as:
 *
 * MSE(batch_of_predictions[0], batch_of_labels[0]) = (1/batchSize) * (batch_of_predictions[0] - batch_of_labels[0])^2
 * MSE(batch_of_predictions[1], batch_of_labels[1]) = (1/batchSize) * (batch_of_predictions[1] - batch_of_labels[1])^2
 * ...
 *
 * So, the number of losses computed is equal to the number of predictions that are made, and each loss back propagates
 * through the associated prediction only.
 */
__global__ void meanSquaredError(
    half *labels, half *predictions, half *elementLoss, half *gradient, uint32_t numElements, bool computeGradient) {
    int element = blockIdx.x * 1024 + (4 * threadIdx.x);

    if (element >= numElements)
        return;

    const half two[2] = {(half)2.0f, (half)2.0f};

    // Always process 4 elements, even when past last element because tensors are always padded to
    // be multiples of 8 bytes (4 half variables) to allow this. This is done for performance reasons.
    double *labels_half4 = (double *)labels;
    double labelsBuffer_half4[1];
    labelsBuffer_half4[0] = labels_half4[element / 4];
    half *labelsBuffer = (half *)labelsBuffer_half4;

    double *predictions_half4 = (double *)predictions;
    double predictionsBuffer_half4[1];
    predictionsBuffer_half4[0] = predictions_half4[element / 4];
    half *predictionsBuffer = (half *)predictionsBuffer_half4;
    half elementLossBuffer[4];
    half gradientBuffer[4];

    half2 buffer0, buffer1;

    buffer0 = __hsub2(((half2 *)labelsBuffer)[0], ((half2 *)predictionsBuffer)[0]);
    ((half2 *)elementLossBuffer)[0] = __hmul2(buffer0, buffer0);
    buffer1 = __hsub2(((half2 *)labelsBuffer)[1], ((half2 *)predictionsBuffer)[1]);
    ((half2 *)elementLossBuffer)[1] = __hmul2(buffer1, buffer1);
    if (computeGradient) {
        ((half2 *)gradientBuffer)[0] = __hmul2(((half2 *)two)[0], buffer0);
        ((half2 *)gradientBuffer)[1] = __hmul2(((half2 *)two)[0], buffer1);
        double *gradientBuffer_half4 = (double *)gradientBuffer;
        double *gradient_half4 = (double *)gradient;
        gradient_half4[element / 4] = gradientBuffer_half4[0];
    }

    double *elementLossBuffer_half4 = (double *)elementLossBuffer;
    double *elementLoss_half4 = (double *)elementLoss;
    elementLoss_half4[element / 4] = elementLossBuffer_half4[0];
}

__global__ void meanSquaredError(
    float *labels, half *predictions, half *elementLoss, half *gradient, uint32_t numElements, bool computeGradient) {
    int element = blockIdx.x * 1024 + (4 * threadIdx.x);

    if (element >= numElements)
        return;

    const half zero[2] = {(half)0.0f, (half)0.0f};
    const half two[2] = {(half)2.0f, (half)2.0f};

    // Always process 4 elements, even when past last element because tensors are always padded to
    // be multiples of 8 bytes (4 half variables) to allow this. This is done for performance reasons.
    float2 *labels_float2 = (float2 *)labels;
    float2 labelsBuffer_float2;
    half labelsBuffer_half4[4];

    labelsBuffer_float2 = labels_float2[element / 2];
    ((half2 *)labelsBuffer_half4)[0] = __float22half2_rn(labelsBuffer_float2);
    if (numElements + 2 >= numElements) {
        ((half2 *)labelsBuffer_half4)[1] = ((half2 *)zero)[0];
    } else {
        labelsBuffer_float2 = labels_float2[(element / 2) + 1];
        ((half2 *)labelsBuffer_half4)[1] = __float22half2_rn(labelsBuffer_float2);
    }
    half *labelsBuffer = (half *)labelsBuffer_half4;

    double *predictions_half4 = (double *)predictions;
    double predictionsBuffer_half4[1];
    predictionsBuffer_half4[0] = predictions_half4[element / 4];
    half *predictionsBuffer = (half *)predictionsBuffer_half4;
    half elementLossBuffer[4];
    half gradientBuffer[4];

    half2 buffer0, buffer1;

    buffer0 = __hsub2(((half2 *)labelsBuffer)[0], ((half2 *)predictionsBuffer)[0]);
    ((half2 *)elementLossBuffer)[0] = __hmul2(buffer0, buffer0);
    buffer1 = __hsub2(((half2 *)labelsBuffer)[1], ((half2 *)predictionsBuffer)[1]);
    ((half2 *)elementLossBuffer)[1] = __hmul2(buffer1, buffer1);
    if (computeGradient) {
        ((half2 *)gradientBuffer)[0] = __hmul2(((half2 *)two)[0], buffer0);
        ((half2 *)gradientBuffer)[1] = __hmul2(((half2 *)two)[0], buffer1);
        double *gradientBuffer_half4 = (double *)gradientBuffer;
        double *gradient_half4 = (double *)gradient;
        gradient_half4[element / 4] = gradientBuffer_half4[0];
    }

    double *elementLossBuffer_half4 = (double *)elementLossBuffer;
    double *elementLoss_half4 = (double *)elementLoss;
    elementLoss_half4[element / 4] = elementLossBuffer_half4[0];
}

__global__ void meanSquaredError(
    float *labels, float *predictions, float *elementLoss, float *gradient, uint32_t numElements, bool computeGradient) {
    int element = blockIdx.x * 1024 + (2 * threadIdx.x);

    if (element >= numElements)
        return;

    float2 *labels_float2 = (float2 *)labels;
    float2 labelsBuffer;

    float2 *predictions_float2 = (float2 *)predictions;
    float2 predictionsBuffer;

    float2 elementLossBuffer;
    float2 *elementLoss_float2 = (float2 *)elementLoss;

    float2 gradientBuffer;
    float2 *gradient_float2 = (float2 *)gradient;

    float buffer0, buffer1;

    labelsBuffer = labels_float2[element / 2];
    predictionsBuffer = predictions_float2[element / 2];

    buffer0 = labelsBuffer.x - predictionsBuffer.x;
    elementLossBuffer.x = buffer0 * buffer0;
    buffer1 = labelsBuffer.y - predictionsBuffer.y;
    elementLossBuffer.y = buffer1 * buffer1;

    if (computeGradient) {
        gradientBuffer.x = 2.0f * buffer0;
        gradientBuffer.y = 2.0f * buffer1;
        gradient_float2[element / 2] = gradientBuffer;
    }

    // Tensors are always padded to be multiples of 8 bytes (4 half variables) to allow this, without the possibility
    // of indexing out of bounds.
    elementLoss_float2[element / 2] = elementLossBuffer;

    element += 512;
    if (element >= numElements)
        return;

    labelsBuffer = labels_float2[element / 2];
    predictionsBuffer = predictions_float2[element / 2];

    buffer0 = labelsBuffer.x - predictionsBuffer.x;
    elementLossBuffer.x = buffer0 * buffer0;
    buffer1 = labelsBuffer.y - predictionsBuffer.y;
    elementLossBuffer.y = buffer1 * buffer1;

    if (computeGradient) {
        gradientBuffer.x = 2.0f * buffer0;
        gradientBuffer.y = 2.0f * buffer1;
        gradient_float2[element / 2] = gradientBuffer;
    }

    // Tensors are always padded to be multiples of 8 bytes (4 half variables) to allow this, without the possibility
    // of indexing out of bounds.
    elementLoss_float2[element / 2] = elementLossBuffer;
}

__global__ void meanSquaredError(
    float *labels, half *predictions, float *elementLoss, half *gradient, uint32_t numElements, bool computeGradient) {
    int element = blockIdx.x * 1024 + (2 * threadIdx.x);

    if (element >= numElements)
        return;

    const half two[2] = {(half)2.0f, (half)2.0f};

    float2 *labels_float2 = (float2 *)labels;
    float2 labelsBuffer;

    half2 *predictions_half2 = (half2 *)predictions;
    half2 predictionsBuffer_half2;
    float2 predictionsBuffer;

    float2 *elementLoss_float2 = (float2 *)elementLoss;
    float2 elementLossBuffer;

    half2 *gradient_half2 = (half2 *)gradient;
    half2 gradientBuffer;

    float2 buffer;

    labelsBuffer = labels_float2[element / 2];
    predictionsBuffer_half2 = predictions_half2[element / 2];
    predictionsBuffer = __half22float2(predictionsBuffer_half2);

    buffer.x = labelsBuffer.x - predictionsBuffer.x;
    elementLossBuffer.x = buffer.x * buffer.x;

    if (element + 1 < numElements) {
        buffer.y = labelsBuffer.y - predictionsBuffer.y;
        elementLossBuffer.y = buffer.y * buffer.y;
    }

    // Tensors are always padded to be multiples of 8 bytes (4 half variables) to allow this, without the possibility
    // of indexing out of bounds.
    elementLoss_float2[element / 2] = elementLossBuffer;

    if (computeGradient) {
        gradientBuffer = __hmul2(((half2 *)two)[0], __float22half2_rn(buffer));
        gradient_half2[element / 2] = gradientBuffer;
    }

    element += 512;
    if (element >= numElements)
        return;

    labelsBuffer = labels_float2[element / 2];
    predictionsBuffer_half2 = predictions_half2[element / 2];
    predictionsBuffer = __half22float2(predictionsBuffer_half2);

    buffer.x = labelsBuffer.x - predictionsBuffer.x;
    elementLossBuffer.x = buffer.x * buffer.x;

    if (element + 1 < numElements) {
        buffer.y = labelsBuffer.y - predictionsBuffer.y;
        elementLossBuffer.y = buffer.y * buffer.y;
    }

    // Tensors are always padded to be multiples of 8 bytes (4 half variables) to allow this, without the possibility
    // of indexing out of bounds.
    elementLoss_float2[element / 2] = elementLossBuffer;

    if (computeGradient) {
        gradientBuffer = __hmul2(((half2 *)two)[0], __float22half2_rn(buffer));
        gradient_half2[element / 2] = gradientBuffer;
    }
}

template <typename LABEL_TYPE, typename PREDICTION_TYPE, typename LOSS_TYPE>
__global__ void meanSquaredError(LABEL_TYPE *labels,
                                 PREDICTION_TYPE *predictions,
                                 LOSS_TYPE *elementLoss,
                                 PREDICTION_TYPE *gradient,
                                 uint32_t numElements,
                                 bool computeGradient) {
    int32_t element = blockIdx.x * 1024 + threadIdx.x;

    LOSS_TYPE buffer;

    if (element >= numElements)
        return;
    buffer = (LOSS_TYPE)(float)labels[element] - (LOSS_TYPE)(float)predictions[element];
    elementLoss[element] = buffer * buffer;
    if (computeGradient)
        gradient[element] = (LOSS_TYPE)2 * buffer;

    element += 256;
    if (element >= numElements)
        return;
    buffer = (LOSS_TYPE)(float)labels[element] - (LOSS_TYPE)(float)predictions[element];
    elementLoss[element] = buffer * buffer;
    if (computeGradient)
        gradient[element] = (LOSS_TYPE)2 * buffer;

    element += 256;
    if (element >= numElements)
        return;
    buffer = (LOSS_TYPE)(float)labels[element] - (LOSS_TYPE)(float)predictions[element];
    elementLoss[element] = buffer * buffer;
    if (computeGradient)
        gradient[element] = (LOSS_TYPE)2 * buffer;

    element += 256;
    if (element >= numElements)
        return;
    buffer = (LOSS_TYPE)(float)labels[element] - (LOSS_TYPE)(float)predictions[element];
    elementLoss[element] = buffer * buffer;
    if (computeGradient)
        gradient[element] = (LOSS_TYPE)2 * buffer;
}

template <typename LABEL_TYPE, typename PREDICTION_TYPE, typename LOSS_TYPE>
void launchMeanSquaredError(void *labels_d,
                            void *predictions_d,
                            void *elementLoss_d,
                            void *gradient_d,
                            uint32_t numPredictions,
                            uint32_t batchSize,
                            Stream stream,
                            bool computeBatchLoss,
                            bool computeGradient) {
    uint32_t numElements = batchSize * numPredictions;

    dim3 blockSize(min(256, numElements));
    dim3 gridSize((numElements + 1023) / 1024);
    ScopedGpu scopedGpu(stream.getGpuNum());

    meanSquaredError<<<gridSize, blockSize, 0, stream>>>((LABEL_TYPE *)labels_d,
                                                         (PREDICTION_TYPE *)predictions_d,
                                                         (LOSS_TYPE *)elementLoss_d,
                                                         (PREDICTION_TYPE *)gradient_d,
                                                         numElements,
                                                         computeGradient);
}

template void launchMeanSquaredError<half, half, half>(void *labels_d,
                                                       void *predictions_d,
                                                       void *elementLoss_d,
                                                       void *gradient,
                                                       uint32_t numPredictions,
                                                       uint32_t batchSize,
                                                       Stream stream,
                                                       bool computeBatchLoss,
                                                       bool computeGradient);

template void launchMeanSquaredError<half, half, float>(void *labels_d,
                                                        void *predictions_d,
                                                        void *elementLoss_d,
                                                        void *gradient,
                                                        uint32_t numPredictions,
                                                        uint32_t batchSize,
                                                        Stream stream,
                                                        bool computeBatchLoss,
                                                        bool computeGradient);

template void launchMeanSquaredError<half, float, half>(void *labels_d,
                                                        void *predictions_d,
                                                        
                                                        void *elementLoss_d,
                                                        void *gradient,
                                                        uint32_t numPredictions,
                                                        uint32_t batchSize,
                                                        Stream stream,
                                                        
                                                        bool computeBatchLoss,
                                                        bool computeGradient);

template void launchMeanSquaredError<half, float, float>(void *labels_d,
                                                         void *predictions_d,
                                                         
                                                         void *elementLoss_d,
                                                         void *gradient,
                                                         uint32_t numPredictions,
                                                         uint32_t batchSize,
                                                         Stream stream,
                                                         
                                                         bool computeBatchLoss,
                                                         bool computeGradient);

template void launchMeanSquaredError<float, half, half>(void *labels_d,
                                                        void *predictions_d,
                                                        
                                                        void *elementLoss_d,
                                                        void *gradient,
                                                        uint32_t numPredictions,
                                                        uint32_t batchSize,
                                                        Stream stream,
                                                        
                                                        bool computeBatchLoss,
                                                        bool computeGradient);

template void launchMeanSquaredError<float, half, float>(void *labels_d,
                                                         void *predictions_d,
                                                         
                                                         void *elementLoss_d,
                                                         void *gradient,
                                                         uint32_t numPredictions,
                                                         uint32_t batchSize,
                                                         Stream stream,
                                                         
                                                         bool computeBatchLoss,
                                                         bool computeGradient);

template void launchMeanSquaredError<float, float, half>(void *labels_d,
                                                         void *predictions_d,
                                                         
                                                         void *elementLoss_d,
                                                         void *gradient,
                                                         uint32_t numPredictions,
                                                         uint32_t batchSize,
                                                         Stream stream,
                                                         
                                                         bool computeBatchLoss,
                                                         bool computeGradient);

template void launchMeanSquaredError<float, float, float>(void *labels_d,
                                                          void *predictions_d,
                                                          
                                                          void *elementLoss_d,
                                                          void *gradient,
                                                          uint32_t numPredictions,
                                                          uint32_t batchSize,
                                                          Stream stream,
                                                          
                                                          bool computeBatchLoss,
                                                          bool computeGradient);

// uint32_t
template void launchMeanSquaredError<uint32_t, half, half>(void *labels_d,
                                                           void *predictions_d,
                                                           
                                                           void *elementLoss_d,
                                                           void *gradient,
                                                           uint32_t numPredictions,
                                                           uint32_t batchSize,
                                                           Stream stream,
                                                           
                                                           bool computeBatchLoss,
                                                           bool computeGradient);

template void launchMeanSquaredError<uint32_t, half, float>(void *labels_d,
                                                            void *predictions_d,
                                                            
                                                            void *elementLoss_d,
                                                            void *gradient,
                                                            uint32_t numPredictions,
                                                            uint32_t batchSize,
                                                            Stream stream,
                                                            
                                                            bool computeBatchLoss,
                                                            bool computeGradient);

template void launchMeanSquaredError<uint32_t, float, half>(void *labels_d,
                                                            void *predictions_d,
                                                            
                                                            void *elementLoss_d,
                                                            void *gradient,
                                                            uint32_t numPredictions,
                                                            uint32_t batchSize,
                                                            Stream stream,
                                                            
                                                            bool computeBatchLoss,
                                                            bool computeGradient);

template void launchMeanSquaredError<uint32_t, float, float>(void *labels_d,
                                                             void *predictions_d,
                                                             
                                                             void *elementLoss_d,
                                                             void *gradient,
                                                             uint32_t numPredictions,
                                                             uint32_t batchSize,
                                                             Stream stream,
                                                             
                                                             bool computeBatchLoss,
                                                             bool computeGradient);

// uint16_t
template void launchMeanSquaredError<uint16_t, half, half>(void *labels_d,
                                                           void *predictions_d,
                                                           
                                                           void *elementLoss_d,
                                                           void *gradient,
                                                           uint32_t numPredictions,
                                                           uint32_t batchSize,
                                                           Stream stream,
                                                           
                                                           bool computeBatchLoss,
                                                           bool computeGradient);

template void launchMeanSquaredError<uint16_t, half, float>(void *labels_d,
                                                            void *predictions_d,
                                                            
                                                            void *elementLoss_d,
                                                            void *gradient,
                                                            uint32_t numPredictions,
                                                            uint32_t batchSize,
                                                            Stream stream,
                                                            
                                                            bool computeBatchLoss,
                                                            bool computeGradient);

template void launchMeanSquaredError<uint16_t, float, half>(void *labels_d,
                                                            void *predictions_d,
                                                            
                                                            void *elementLoss_d,
                                                            void *gradient,
                                                            uint32_t numPredictions,
                                                            uint32_t batchSize,
                                                            Stream stream,
                                                            
                                                            bool computeBatchLoss,
                                                            bool computeGradient);

template void launchMeanSquaredError<uint16_t, float, float>(void *labels_d,
                                                             void *predictions_d,
                                                             
                                                             void *elementLoss_d,
                                                             void *gradient,
                                                             uint32_t numPredictions,
                                                             uint32_t batchSize,
                                                             Stream stream,
                                                             
                                                             bool computeBatchLoss,
                                                             bool computeGradient);

// uint8_t
template void launchMeanSquaredError<uint8_t, half, half>(void *labels_d,
                                                          void *predictions_d,
                                                          
                                                          void *elementLoss_d,
                                                          void *gradient,
                                                          uint32_t numPredictions,
                                                          uint32_t batchSize,
                                                          Stream stream,
                                                          
                                                          bool computeBatchLoss,
                                                          bool computeGradient);

template void launchMeanSquaredError<uint8_t, half, float>(void *labels_d,
                                                           void *predictions_d,
                                                           
                                                           void *elementLoss_d,
                                                           void *gradient,
                                                           uint32_t numPredictions,
                                                           uint32_t batchSize,
                                                           Stream stream,
                                                           
                                                           bool computeBatchLoss,
                                                           bool computeGradient);

template void launchMeanSquaredError<uint8_t, float, half>(void *labels_d,
                                                           void *predictions_d,
                                                           
                                                           void *elementLoss_d,
                                                           void *gradient,
                                                           uint32_t numPredictions,
                                                           uint32_t batchSize,
                                                           Stream stream,
                                                           
                                                           bool computeBatchLoss,
                                                           bool computeGradient);

template void launchMeanSquaredError<uint8_t, float, float>(void *labels_d,
                                                            void *predictions_d,
                                                            
                                                            void *elementLoss_d,
                                                            void *gradient,
                                                            uint32_t numPredictions,
                                                            uint32_t batchSize,
                                                            Stream stream,
                                                            
                                                            bool computeBatchLoss,
                                                            bool computeGradient);

// int32_t
template void launchMeanSquaredError<int32_t, half, half>(void *labels_d,
                                                          void *predictions_d,
                                                          
                                                          void *elementLoss_d,
                                                          void *gradient,
                                                          uint32_t numPredictions,
                                                          uint32_t batchSize,
                                                          Stream stream,
                                                          
                                                          bool computeBatchLoss,
                                                          bool computeGradient);

template void launchMeanSquaredError<int32_t, half, float>(void *labels_d,
                                                           void *predictions_d,
                                                           
                                                           void *elementLoss_d,
                                                           void *gradient,
                                                           uint32_t numPredictions,
                                                           uint32_t batchSize,
                                                           Stream stream,
                                                           
                                                           bool computeBatchLoss,
                                                           bool computeGradient);

template void launchMeanSquaredError<int32_t, float, half>(void *labels_d,
                                                           void *predictions_d,
                                                           
                                                           void *elementLoss_d,
                                                           void *gradient,
                                                           uint32_t numPredictions,
                                                           uint32_t batchSize,
                                                           Stream stream,
                                                           
                                                           bool computeBatchLoss,
                                                           bool computeGradient);

template void launchMeanSquaredError<int32_t, float, float>(void *labels_d,
                                                            void *predictions_d,
                                                            
                                                            void *elementLoss_d,
                                                            void *gradient,
                                                            uint32_t numPredictions,
                                                            uint32_t batchSize,
                                                            Stream stream,
                                                            
                                                            bool computeBatchLoss,
                                                            bool computeGradient);

// int16_t
template void launchMeanSquaredError<int16_t, half, half>(void *labels_d,
                                                          void *predictions_d,
                                                          
                                                          void *elementLoss_d,
                                                          void *gradient,
                                                          uint32_t numPredictions,
                                                          uint32_t batchSize,
                                                          Stream stream,
                                                          
                                                          bool computeBatchLoss,
                                                          bool computeGradient);

template void launchMeanSquaredError<int16_t, half, float>(void *labels_d,
                                                           void *predictions_d,
                                                           
                                                           void *elementLoss_d,
                                                           void *gradient,
                                                           uint32_t numPredictions,
                                                           uint32_t batchSize,
                                                           Stream stream,
                                                           
                                                           bool computeBatchLoss,
                                                           bool computeGradient);

template void launchMeanSquaredError<int16_t, float, half>(void *labels_d,
                                                           void *predictions_d,
                                                           
                                                           void *elementLoss_d,
                                                           void *gradient,
                                                           uint32_t numPredictions,
                                                           uint32_t batchSize,
                                                           Stream stream,
                                                           
                                                           bool computeBatchLoss,
                                                           bool computeGradient);

template void launchMeanSquaredError<int16_t, float, float>(void *labels_d,
                                                            void *predictions_d,
                                                            
                                                            void *elementLoss_d,
                                                            void *gradient,
                                                            uint32_t numPredictions,
                                                            uint32_t batchSize,
                                                            Stream stream,
                                                            
                                                            bool computeBatchLoss,
                                                            bool computeGradient);

// int8_t
template void launchMeanSquaredError<int8_t, half, half>(void *labels_d,
                                                         void *predictions_d,
                                                         
                                                         void *elementLoss_d,
                                                         void *gradient,
                                                         uint32_t numPredictions,
                                                         uint32_t batchSize,
                                                         Stream stream,
                                                         
                                                         bool computeBatchLoss,
                                                         bool computeGradient);

template void launchMeanSquaredError<int8_t, half, float>(void *labels_d,
                                                          void *predictions_d,
                                                          
                                                          void *elementLoss_d,
                                                          void *gradient,
                                                          uint32_t numPredictions,
                                                          uint32_t batchSize,
                                                          Stream stream,
                                                          
                                                          bool computeBatchLoss,
                                                          bool computeGradient);

template void launchMeanSquaredError<int8_t, float, half>(void *labels_d,
                                                          void *predictions_d,
                                                          
                                                          void *elementLoss_d,
                                                          void *gradient,
                                                          uint32_t numPredictions,
                                                          uint32_t batchSize,
                                                          Stream stream,
                                                          
                                                          bool computeBatchLoss,
                                                          bool computeGradient);

template void launchMeanSquaredError<int8_t, float, float>(void *labels_d,
                                                           void *predictions_d,
                                                           
                                                           void *elementLoss_d,
                                                           void *gradient,
                                                           uint32_t numPredictions,
                                                           uint32_t batchSize,
                                                           Stream stream,
                                                           
                                                           bool computeBatchLoss,
                                                           bool computeGradient);

// bool
template void launchMeanSquaredError<bool, half, half>(void *labels_d,
                                                       void *predictions_d,
                                                       
                                                       void *elementLoss_d,
                                                       void *gradient,
                                                       uint32_t numPredictions,
                                                       uint32_t batchSize,
                                                       Stream stream,
                                                       
                                                       bool computeBatchLoss,
                                                       bool computeGradient);

template void launchMeanSquaredError<bool, half, float>(void *labels_d,
                                                        void *predictions_d,
                                                        
                                                        void *elementLoss_d,
                                                        void *gradient,
                                                        uint32_t numPredictions,
                                                        uint32_t batchSize,
                                                        Stream stream,
                                                        
                                                        bool computeBatchLoss,
                                                        bool computeGradient);

template void launchMeanSquaredError<bool, float, half>(void *labels_d,
                                                        void *predictions_d,
                                                        
                                                        void *elementLoss_d,
                                                        void *gradient,
                                                        uint32_t numPredictions,
                                                        uint32_t batchSize,
                                                        Stream stream,
                                                        
                                                        bool computeBatchLoss,
                                                        bool computeGradient);

template void launchMeanSquaredError<bool, float, float>(void *labels_d,
                                                         void *predictions_d,
                                                         
                                                         void *elementLoss_d,
                                                         void *gradient,
                                                         uint32_t numPredictions,
                                                         uint32_t batchSize,
                                                         Stream stream,
                                                         
                                                         bool computeBatchLoss,
                                                         bool computeGradient);
